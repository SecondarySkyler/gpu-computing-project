#include "hip/hip_runtime.h"
#include "../include/transpose_csr_v2.cuh"
#include "../include/utilities.cuh"

__global__ void count_nnz(int nnz, int* csrColumnIndices, int* cscColPtr) {
    __shared__ int shared[BLOCK_SIZE];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    shared[threadIdx.x] = csrColumnIndices[tid];

    __syncthreads();
    if (tid < nnz) {
        // int col = csrColumnIndices[tid];
        int col = shared[threadIdx.x];
        atomicAdd(&cscColPtr[col + 1], 1);
    }
}

__global__ void scanLA(int* d_cscColPtr, int* d_auxBlockSums, int cols) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    __shared__ int block[BLOCK_SIZE];
    int i = bid * BLOCK_SIZE + tid;
    if (i < cols) {
        block[tid] = d_cscColPtr[i];
    } else {
        block[tid] = 0;
    }
    __syncthreads();

    for (int stride = 1; stride < BLOCK_SIZE; stride *= 2) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index < BLOCK_SIZE) {
            block[index] += block[index - stride];
        }
        __syncthreads();
    }

    for (int stride = BLOCK_SIZE / 4; stride > 0; stride /= 2) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index + stride < BLOCK_SIZE) {
            block[index + stride] += block[index];
        }
        __syncthreads();
    }

    if (i < cols) {
        d_cscColPtr[i] = block[tid];
    }
    if (tid == BLOCK_SIZE - 1) {
        d_auxBlockSums[bid] = block[tid];
    }

}

__global__ void uniformUpdate(int* d_cscColPtr, int* d_auxBlockSums, int cols) {
    int bid = blockIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bid > 0) {
        d_cscColPtr[idx] += d_auxBlockSums[bid - 1];
    }
}

__global__ void fillCSC_ds(int num_rows, int* col_offsets, const int* row_offsets, const int* col_indices, const dtype* values, dtype* csc_values, int* csc_row_indices) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        for (int j = row_offsets[row]; j < row_offsets[row + 1]; j++) {
            int col = col_indices[j];
            int index = atomicAdd(&col_offsets[col], 1);
            csc_values[index] = values[j];
            csc_row_indices[index] = row;
        }
    }
}

void transpose_CSR_v2(std::string fileName) {
    int rows, cols, nnz;
    int* csrRowPointers, *csrColumnIndices;
    dtype* csrValues;

    parseCsvToCsr(rows, cols, nnz, csrRowPointers, csrColumnIndices, csrValues, fileName);

    // Declare CSC matrix variables
    int* cscColPtr = (int*)malloc((cols + 1) * sizeof(int));
    int* cscRowIdx = (int*)malloc(nnz * sizeof(int));
    dtype* cscVal = (dtype*)malloc(nnz * sizeof(dtype));

    int* cscColPtrCollector = (int*)malloc((cols + 1) * sizeof(int)); // used to collect the results from the device

    // Initialize cscColPtr with zeros
    for (int i = 0; i < cols + 1; i++) {
        cscColPtr[i] = 0;
        cscColPtrCollector[i] = 0;
    }

    // -- Device memory allocation --
    int* d_csrRowPointers;
    int* d_csrColumnIndices;
    dtype* d_csrValues;
    int* d_cscColPtr;
    int* d_cscRowIdx;
    dtype* d_cscVal;

    // Allocate device memory
    CHECK(hipMalloc(&d_csrRowPointers, (rows + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_csrColumnIndices, nnz * sizeof(int)));
    CHECK(hipMalloc(&d_csrValues, nnz * sizeof(dtype)));
    CHECK(hipMalloc(&d_cscColPtr, (cols + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_cscRowIdx, nnz * sizeof(int)));
    CHECK(hipMalloc(&d_cscVal, nnz * sizeof(dtype)));

    // Copy data to device memory
    CHECK(hipMemcpy(d_csrRowPointers, csrRowPointers, (rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_csrColumnIndices, csrColumnIndices, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_csrValues, csrValues, nnz * sizeof(dtype), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_cscColPtr, cscColPtr, (cols + 1) * sizeof(int), hipMemcpyHostToDevice));

    // Create cuda events to measure the time
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    float step1_ms = 0.0; // Count
    float step2_ms = 0.0; // Scan
    float step3_ms = 0.0; // Uniform update
    float step4_ms = 0.0; // Fill CSC

    // Warm up the GPU
    warm_up_gpu();

    /**
     * Step 1: Count the number of non-zero elements in each column
     * First I execute once the kernel
     * Then I measure the time of 101 executions
     * This is done because the execution of the kernel alters the data in the device memory
     */
    int gridSize = (nnz + BLOCK_SIZE - 1) / BLOCK_SIZE;

    count_nnz<<<gridSize, BLOCK_SIZE>>>(nnz, d_csrColumnIndices, d_cscColPtr);
    
    // Copy the result to the host
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(cscColPtrCollector, d_cscColPtr, (cols + 1) * sizeof(int), hipMemcpyDeviceToHost));

    // Measure the time of 100 executions
    CHECK(hipEventRecord(start, 0));
    for (int i = 0; i < NUM_REPS; i++) {
        count_nnz<<<gridSize, BLOCK_SIZE>>>(nnz, d_csrColumnIndices, d_cscColPtr);
    }
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&step1_ms, start, stop));
    CHECK(hipDeviceSynchronize());
    printf("Step 1: %f ms\n", step1_ms);
    
    /**
     * Step 2: Perform exclusive scan
     * Same as before, first I execute once the kernel
     * Then I measure the time of 100 executions
     */
    gridSize = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int* aux = (int*)malloc(gridSize * sizeof(int));
    int* d_auxBlockSums;
    CHECK(hipMalloc(&d_auxBlockSums, gridSize * sizeof(int)));
    CHECK(hipMemcpy(d_cscColPtr, cscColPtrCollector, (cols + 1) * sizeof(int), hipMemcpyHostToDevice)); // this should "reset" the data in the device memory
    scanLA<<<gridSize, BLOCK_SIZE>>>(d_cscColPtr, d_auxBlockSums, cols + 1);

    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(aux, d_auxBlockSums, gridSize * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cscColPtrCollector, d_cscColPtr, (cols + 1) * sizeof(int), hipMemcpyDeviceToHost));

    // Measure the time of 100 executions
    CHECK(hipEventRecord(start, 0));
    for (int i = 0; i < NUM_REPS; i++) {
        scanLA<<<gridSize, BLOCK_SIZE>>>(d_cscColPtr, d_auxBlockSums, cols + 1);
    }
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&step2_ms, start, stop));
    CHECK(hipDeviceSynchronize());
    printf("Step 2: %f ms\n", step2_ms);

    // Compute the scan of the aux array and copy it to the device
    for (int i = 1; i < gridSize; i++) {
        aux[i] += aux[i - 1];
    }

    CHECK(hipMemcpy(d_auxBlockSums, aux, gridSize * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_cscColPtr, cscColPtrCollector, (cols + 1) * sizeof(int), hipMemcpyHostToDevice));

    uniformUpdate<<<gridSize, BLOCK_SIZE>>>(d_cscColPtr, d_auxBlockSums, cols + 1);

    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(cscColPtrCollector, d_cscColPtr, (cols + 1) * sizeof(int), hipMemcpyDeviceToHost));

    // Measure the time of 100 executions
    CHECK(hipEventRecord(start, 0));
    for (int i = 0; i < NUM_REPS; i++) {
        uniformUpdate<<<gridSize, BLOCK_SIZE>>>(d_cscColPtr, d_auxBlockSums, cols + 1);
    }
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&step3_ms, start, stop));
    CHECK(hipDeviceSynchronize());
    printf("Step 3: %f ms\n", step3_ms);

    /**
     * Step 3: Compute the cscRowIdx and cscVal
     */
    CHECK(hipMemcpy(d_cscColPtr, cscColPtrCollector, (cols + 1) * sizeof(int), hipMemcpyHostToDevice));
    gridSize = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    fillCSC_ds<<<gridSize, BLOCK_SIZE>>>(
        rows, d_cscColPtr, d_csrRowPointers, d_csrColumnIndices, d_csrValues, d_cscVal, d_cscRowIdx
    );

    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(cscRowIdx, d_cscRowIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cscVal, d_cscVal, nnz * sizeof(dtype), hipMemcpyDeviceToHost));

    // Measure the time of 100 executions
    CHECK(hipEventRecord(start, 0));
    for (int i = 0; i < NUM_REPS; i++) {
        fillCSC_ds<<<gridSize, BLOCK_SIZE>>>(
            rows, d_cscColPtr, d_csrRowPointers, d_csrColumnIndices, d_csrValues, d_cscVal, d_cscRowIdx
        );
    }
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&step4_ms, start, stop));
    CHECK(hipDeviceSynchronize());
    printf("Step 4: %f ms\n", step4_ms);


    // Check if the result is correct
    dtype* groundTruth = generateGroundTruthFromMTX(fileName);

    if (checkResultCSR(groundTruth, cscColPtrCollector, cscRowIdx, cscVal, rows, cols)) {
        printf("Performed CSR transposition on matrix %s\n", fileName.c_str());
        // printf("Bandwidth: %f GB/s\n", 4 * nnz * sizeof(int) * 1e-6 * NUM_REPS / milliseconds);
        printf("Status: Correct\n");
        printf("--------------------------------\n");
    } else {
        printf("The result is incorrect\n");
        printf("--------------------------------\n");
    }

    // Free memory
    CHECK(hipFree(d_csrRowPointers));
    CHECK(hipFree(d_csrColumnIndices));
    CHECK(hipFree(d_csrValues));
    CHECK(hipFree(d_cscColPtr));
    CHECK(hipFree(d_cscRowIdx));
    CHECK(hipFree(d_cscVal));
    CHECK(hipFree(d_auxBlockSums));

    free(csrRowPointers);
    free(csrColumnIndices);
    free(csrValues);
    free(cscColPtr);
    free(cscRowIdx);
    free(cscVal);
    free(cscColPtrCollector);
    free(groundTruth);
    free(aux);
}
