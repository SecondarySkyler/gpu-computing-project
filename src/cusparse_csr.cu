#include "../include/cusparse_csr.cuh"
#include "../include/utilities.cuh"

void cusparse_CSR(std::string fileName) {
    int rows, cols, nnz;
    int *csrRowPointers, *csrColumnIndices;
    dtype *csrValues;

    parseCsvToCsr(rows, cols, nnz, csrRowPointers, csrColumnIndices, csrValues, fileName);

    // Host memory management
    int* cscRowIndices = (int*)malloc(nnz * sizeof(int));
    int* cscColumnPointers = (int*)malloc((cols + 1) * sizeof(int));
    dtype* cscValues = (dtype*)malloc(nnz * sizeof(dtype));

    // Device memory management
    int *d_csrRowPointers, *d_csrColumnIndices;
    dtype *d_csrValues;

    int *d_cscRowIndices, *d_cscColumnPointers;
    dtype *d_cscValues;

    CHECK(hipMalloc(&d_csrRowPointers, (rows + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_csrColumnIndices, nnz * sizeof(int)));
    CHECK(hipMalloc(&d_csrValues, nnz * sizeof(dtype)));
    CHECK(hipMalloc(&d_cscRowIndices, nnz * sizeof(int)));
    CHECK(hipMalloc(&d_cscColumnPointers, (cols + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_cscValues, nnz * sizeof(dtype)));

    CHECK(hipMemcpy(d_csrRowPointers, csrRowPointers, (rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_csrColumnIndices, csrColumnIndices, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_csrValues, csrValues, nnz * sizeof(dtype), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    float milliseconds = 0.0;

    // cuSparse APIs
    hipsparseHandle_t handle;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Reserve buffer
    CHECK_CUSPARSE(
        hipsparseCsr2cscEx2_bufferSize(
            handle,
            rows,
            cols,
            nnz,
            d_csrValues,
            d_csrRowPointers,
            d_csrColumnIndices,
            d_cscValues,
            d_cscColumnPointers,
            d_cscRowIndices,
            HIP_R_64F, // maybe change to HIP_R_64F
            HIPSPARSE_ACTION_NUMERIC,
            HIPSPARSE_INDEX_BASE_ZERO,
            HIPSPARSE_CSR2CSC_ALG1,
            &bufferSize
        )
    );

    CHECK(hipMalloc(&dBuffer, bufferSize));

    // Convert CSR to CSC
    CHECK(hipEventRecord(start));
    for (int i = 0; i < NUM_REPS; i++) {
        CHECK_CUSPARSE(
            hipsparseCsr2cscEx2(
                handle,
                rows,
                cols,
                nnz,
                d_csrValues,
                d_csrRowPointers,
                d_csrColumnIndices,
                d_cscValues,
                d_cscColumnPointers,
                d_cscRowIndices,
                HIP_R_64F, // maybe change to HIP_R_64F
                HIPSPARSE_ACTION_NUMERIC,
                HIPSPARSE_INDEX_BASE_ZERO,
                HIPSPARSE_CSR2CSC_ALG1,
                dBuffer
            )
        );
    }
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy results back to host
    CHECK(hipMemcpy(cscRowIndices, d_cscRowIndices, nnz * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cscColumnPointers, d_cscColumnPointers, (cols + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cscValues, d_cscValues, nnz * sizeof(dtype), hipMemcpyDeviceToHost));

    // Print results
    // printf("cscRowIndices: ");
    // for (int i = 0; i < nnz; i++) {
    //     printf("%d ", cscRowIndices[i]);
    // }
    // printf("\n");

    // printf("cscColumnPointers: ");
    // for (int i = 0; i < cols + 1; i++) {
    //     printf("%d ", cscColumnPointers[i]);
    // }
    // printf("\n");

    // printf("cscValues: ");
    // for (int i = 0; i < nnz; i++) {
    //     printf("%f ", cscValues[i]);
    // }
    // printf("\n");

    dtype* groundTruth = generateGroundTruthFromMTX(fileName);

    int copy_step_1 = 2 * nnz * sizeof(int); 
    int count_step_1 = 3 * nnz * sizeof(int);
    int scan_step_2 = 2 * (cols + 1) * sizeof(int);
    int index_step_3 = 2 * nnz * sizeof(int);
    int fill_step_3 = 2 * nnz * sizeof(int) + 2 * nnz * sizeof(dtype);

    double total_data = copy_step_1 + count_step_1 + scan_step_2 + index_step_3 + fill_step_3;

    printf("Performed CSR to CSC conversion on matrix %s\n", fileName.c_str());
    if (checkResultCSR(groundTruth, cscColumnPointers, cscRowIndices, cscValues, rows, cols)) {
        printf("Bandwidth: %f GB/s\n", total_data * 1e-6 * NUM_REPS / milliseconds);
        printf("Status: ");
        // green color
        printf("\033[1;32m");
        printf("Correct\n");
        printf("\033[0m");
        printf("--------------------------------\n");
    } else {
        printf("Status: ");
        // red color
        printf("\033[1;31m");
        printf("Incorrect\n");
        printf("\033[0m");
        printf("--------------------------------\n");
    }


    // Free device memory
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK(hipFree(d_csrRowPointers));
    CHECK(hipFree(d_csrColumnIndices));
    CHECK(hipFree(d_csrValues));
    CHECK(hipFree(d_cscRowIndices));
    CHECK(hipFree(d_cscColumnPointers));
    CHECK(hipFree(d_cscValues));

    // Free host memory
    free(csrRowPointers);
    free(csrColumnIndices);
    free(csrValues);
    free(cscRowIndices);
    free(cscColumnPointers);
    free(cscValues);

}