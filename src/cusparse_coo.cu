#include "../include/cusparse_coo.cuh"
#include "../include/utilities.cuh"

void cusparse_COO(std::string fileName) {
    int rows, cols, nnz;
    int* row, *col;
    cutype* val;

    parseCsvToCoo(rows, cols, nnz, row, col, val, fileName);
    cutype* groundTruth = generateCOOGroundTruth(rows, cols, nnz, row, col, val);

    int size = rows * cols;

    cutype* h_identityMatrix = (cutype*)malloc(size * sizeof(cutype));
    cutype* h_result = (cutype*)malloc(size * sizeof(cutype));

    for (int i = 0; i < size; i++) {
        h_result[i] = 0.0;
    }

    // Fill the identity matrix
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (i == j) {
                h_identityMatrix[j * rows + i] = 1.0;
            } else {
                h_identityMatrix[j * rows + i] = 0.0;
            }
        }
    }

    float alpha = 1.0;
    float beta = 0.0;

    // Device memory management
    int *d_row, *d_col;
    cutype *d_val, *d_identityMatrix, *d_result;

    CHECK(hipMalloc(&d_row, nnz * sizeof(int)));
    CHECK(hipMalloc(&d_col, nnz * sizeof(int)));
    CHECK(hipMalloc(&d_val, nnz * sizeof(cutype)));
    CHECK(hipMalloc(&d_identityMatrix, size * sizeof(cutype)));
    CHECK(hipMalloc(&d_result, size * sizeof(cutype)));

    CHECK(hipMemcpy(d_row, row, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_col, col, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_val, val, nnz * sizeof(cutype), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_identityMatrix, h_identityMatrix, size * sizeof(cutype), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_result, h_result, size * sizeof(cutype), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    float milliseconds = 0.0;

    // cuSparse APIs
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t sparseMatrix;
    hipsparseDnMatDescr_t identityMatrix, resultMatrix;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Create sparse matrix
    CHECK_CUSPARSE(
        hipsparseCreateCoo(
            &sparseMatrix, rows, cols, nnz,
            d_row, d_col, d_val,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F
        )
    );

    // Create identity matrix
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(
            &identityMatrix, rows, cols, rows,
            d_identityMatrix, HIP_R_32F, HIPSPARSE_ORDER_COL
        )
    );

    // Create result matrix
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(
            &resultMatrix, rows, cols, rows,
            d_result, HIP_R_32F, HIPSPARSE_ORDER_COL
        )
    );

    // Allocate buffer
    /**
     * This is only needed when the CUSPARSE_ALG requires additional memory
     * Indeed, only the HIPSPARSE_SPMM_COO_ALG2 requires additional memory
     */
    CHECK_CUSPARSE(
        hipsparseSpMM_bufferSize(
            handle, 
            HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, sparseMatrix, identityMatrix, &beta, resultMatrix,
            HIP_R_32F, HIPSPARSE_SPMM_COO_ALG1, &bufferSize
        )
    );

    CHECK(hipMalloc(&dBuffer, bufferSize));

    warm_up_gpu();

    // Execute SpMM
    CHECK(hipEventRecord(start));
    for (int i = 0; i < NUM_REPS; i++) {
        CHECK_CUSPARSE(
            hipsparseSpMM(
                handle, 
                HIPSPARSE_OPERATION_TRANSPOSE,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, sparseMatrix, identityMatrix, &beta, resultMatrix,
                HIP_R_32F, HIPSPARSE_SPMM_COO_ALG1, dBuffer
            )
        );
    }
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    CHECK_CUSPARSE(hipsparseDestroySpMat(sparseMatrix));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(identityMatrix));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(resultMatrix));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // Copy the result back to host
    CHECK(hipMemcpy(h_result, d_result, size * sizeof(cutype), hipMemcpyDeviceToHost));

    // Compare the results
    bool isCorrect = true;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (h_result[i + j * rows] != groundTruth[j + i * rows]) {
                isCorrect = false;
                break;
            }
        }
    }

    // Print the result
    // printf("Result:\n");
    // for (int i = 0; i < rows; i++) {
    //     for (int j = 0; j < cols; j++) {
    //         printf("%f ", h_result[i + j * rows]);
    //     }
    //     printf("\n");
    // }

    // printf("Ground Truth:\n");
    // for (int i = 0; i < rows; i++) {
    //     for (int j = 0; j < cols; j++) {
    //         printf("%f ", groundTruth[j + i * rows]);
    //     }
    //     printf("\n");
    // }
    cutype totalData = (rows * cols) * sizeof(cutype);
    totalData *= 3;

    printf("Performed cuSparse transposition on matrix %s\n", fileName.c_str());
    if (isCorrect) {
        printf("Bandwidth: %f GB/s\n", totalData * 1e-6 * NUM_REPS / milliseconds);
        printf("Status: ");
        // green color
        printf("\033[1;32m");
        printf("Correct\n");
        printf("\033[0m");
        printf("--------------------------------\n");
  } else {
        printf("Status: ");
        // red color
        printf("\033[1;31m");
        printf("Incorrect\n");
        printf("\033[0m");
        printf("--------------------------------\n");
  }


    free(row);
    free(col);
    free(val);
    free(h_identityMatrix);
    free(h_result);
}